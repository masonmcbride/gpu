#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void add_arrays_gpu(int *a, int *b, int *c, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}

void generate_random_array(int *array, int N) {
    for (int i = 0; i < N; i++) {
        array[i] = rand() % 100;
    }
}

int main() {
    int N;
    printf("Enter the size of the arrays: ");
    scanf("%d", &N);

    int *a = (int*)malloc(N * sizeof(int));
    int *b = (int*)malloc(N * sizeof(int));
    int *c = (int*)malloc(N * sizeof(int));

    generate_random_array(a, N);
    generate_random_array(b, N);

    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, N * sizeof(int));
    hipMalloc((void**)&d_b, N * sizeof(int));
    hipMalloc((void**)&d_c, N * sizeof(int));

    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    add_arrays_gpu<<<numBlocks, blockSize>>>(d_a, d_b, d_c, N);
    hipEventRecord(stop);

    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("GPU time: %f milliseconds\n", milliseconds);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);
    return 0;
}